#pragma once
#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <random>
#include <chrono>
#include <cassert>
#include <hip/hip_fp16.h>
#include <algorithm>
#include <utility>

using T = float;
using Thalf = __half;

#define measurelc(x,counter) { \
	auto now = std::chrono::high_resolution_clock::now();\
	auto count##counter = std::chrono::duration<double, std::milli>(now - last).count();\
	printf("%s: %s: %.3lfms\n", __func__, x, count##counter);\
	counter += count##counter;\
	last = now;\
} 
#define measurec(x,counter) { \
	auto now = std::chrono::high_resolution_clock::now();\
	auto count##counter = std::chrono::duration<double, std::milli>(now - last).count();\
	counter += count##counter;\
	last = now;\
} 
#define measurel(x) { \
	auto now = std::chrono::high_resolution_clock::now();\
	printf("%s: %s: %.3lfms\n", __func__, x, std::chrono::duration<double, std::milli>(now - last).count());\
	last = now;\
} 
#define checkCudaFail() {\
	hipError_t err = hipGetLastError();\
	if(err != hipSuccess) {\
		printf("%s: CUDA kernel failed: %s\n", __func__, hipGetErrorString(err));\
		return EXIT_FAILURE;\
	}\
}

template <class T> bool fp_equal(T a, T b) {
	return fabs((float)a - (float)b) < 1e-3;
}

template <class T> class Tensor {
	T *x, *d_x;
	int *counter;
	int N, size;
public:
	Tensor() = delete;
	Tensor(int _N) { 
		N = _N;
		size = sizeof(T) * N;
		x = (T*)malloc(size), d_x = nullptr;
		hipError_t err = hipMalloc((void**)&d_x, size);
		counter = (int*)malloc(sizeof(int));
		*counter = 1;
		if(err != hipSuccess) {
			d_x = nullptr;
			printf("CUDA malloc fail: N=%d\n", N);
		} else { 
			// printf("Tensor malloc(%d) sizeof(T)=%d, %p %p, %p\n", N, (int)sizeof(T), x, d_x, counter);
		}
	}
	Tensor<T>(const Tensor <T> &other) {
		x = other.x, d_x = other.d_x;
		counter = other.counter, (*counter)++;
		N = other.N, size = other.size;
		// printf("copy %p %p %p counter=%d\n", x, d_x, counter, *counter);
	}
	Tensor<T>& operator=(Tensor<T>&& other)
	{
		x = other.x, d_x = other.d_x;
		counter = other.counter, counter++;
		N = other.N, size = other.size;
		return *this;
	}
	Tensor<T>& operator=(const Tensor<T>& other)
	{
		x = other.x, d_x = other.d_x;
		counter = other.counter, counter++;
		N = other.N, size = other.size;
		return *this;
	}

	~Tensor() { 
		if(--*counter == 0) {
			// printf("free: deconstruct %p, %p, %p, counter=%d\n", x, d_x, counter, *counter);
			free(x); hipFree(d_x); 
			free(counter);
		}
	}
	Tensor& rdrange(double l, double r) {
		std::mt19937 eng(618);
		for(int i = 0; i < N; ++i) x[i] = (T)std::uniform_real_distribution<double>(l,l)(eng);
		return *this;
	}
	Tensor& rd01() { return rdrange(0,1); }
	Tensor& todevice() {
		hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
		return *this;
	} 
	Tensor& tohost() {
		hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
		return *this;
	}
	T* d() { return d_x; } // device
	T* h() { return x; }   // host
	// T& operator [] (int i) { return x[i]; }
};

